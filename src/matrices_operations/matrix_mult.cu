//
// Created by henri on 09/01/2020.
//


#include "kernels/kernel_mat_op.hh"



double* mat_mult(double* A, double* B, int NA, int MA, int NB, int MB)
{
    if (MA != NB) {
        std::cerr << "shape error" << std::endl;
        return nullptr;
    }
    hipError_t rc = hipSuccess;

    int SIZE_A = NA*MA;
    int SIZE_B = NB*MB;
    int SIZE_C = NA*MB;


    // Allocate memory on the device
    double* d_A;
    double* d_B;
    double* d_C;
    auto* C = (double*)malloc(SIZE_C * sizeof(double));

    hipMalloc(&d_A, SIZE_A * sizeof(double));
    hipMalloc(&d_B, SIZE_B * sizeof(double));
    hipMalloc(&d_C, SIZE_C * sizeof(double));


    // Copy to device
    rc = hipMemcpy(d_A, &A[0], SIZE_A * sizeof(double), hipMemcpyHostToDevice);
    if (rc)
        std::cout << "error memcpy\n";
    hipMemcpy(d_B, &B[0], SIZE_B * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, SIZE_C * sizeof(double));

    // call the kernel
    matrixMultiplication(d_A, d_B, d_C, NA, MA, NB, MB);
    hipDeviceSynchronize();

    // copy memory back to host
    hipMemcpy(&C[0], d_C, SIZE_C * sizeof(double), hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return C;
}