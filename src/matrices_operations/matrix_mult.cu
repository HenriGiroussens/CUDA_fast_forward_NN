//
// Created by henri on 09/01/2020.
//


#include "kernels/kernel_mat_op.hh"



double* mat_mult(double* A, double* B, int NA, int MA, int NB, int MB)
{
    if (MA != NB) {
        std::cerr << "shape error" << std::endl;
        return nullptr;
    }
    hipError_t rc = hipSuccess;

    int SIZE_A = NA*MA;
    int SIZE_B = NB*MB;
    int SIZE_C = NA*MB;


    // Allocate memory on the device
    double* d_A;
    double* d_B;
    double* d_C;
    auto* C = (double*)malloc(SIZE_C * sizeof(double));

    hipMalloc(&d_A, SIZE_A * sizeof(double));
    hipMalloc(&d_B, SIZE_B * sizeof(double));
    hipMalloc(&d_C, SIZE_C * sizeof(double));


    // Copy to device
    rc = hipMemcpy(d_A, &A[0], SIZE_A * sizeof(double), hipMemcpyHostToDevice);
    if (rc)
        std::cout << "error memcpy\n";
    rc = hipMemcpy(d_B, &B[0], SIZE_B * sizeof(double), hipMemcpyHostToDevice);
    if (rc)
        std::cout << "error memcpy\n";
    rc = hipMemset(d_C, 0, SIZE_C * sizeof(double));
    if (rc)
        std::cout << "error memset\n";

    // call the kernel
    matrixMultiplication(d_A, d_B, d_C, NA, MA, NB, MB);
    hipDeviceSynchronize();

    // copy memory back to host
    rc = hipMemcpy(&C[0], d_C, SIZE_C * sizeof(double), hipMemcpyDeviceToHost);
    if (rc)
        fprintf(stderr,"GPUassert: %s \n", hipGetErrorString(rc));
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return C;
}