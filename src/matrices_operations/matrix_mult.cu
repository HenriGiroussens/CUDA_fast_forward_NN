//
// Created by henri on 09/01/2020.
//


#include "kernels/kernel_mat_op.hh"



float* mat_mult(float* A, float* B, int NA, int MA, int NB, int MB)
{
    if (MA != NB) {
        std::cerr << "shape error" << std::endl;
        return nullptr;
    }
    hipError_t rc = hipSuccess;

    int SIZE_A = NA*MA;
    int SIZE_B = NB*MB;
    int SIZE_C = NA*MB;


    // Allocate memory on the device
    float* d_A;
    float* d_B;
    float* d_C;
    auto* C = (float*)malloc(SIZE_C * sizeof(float));

    hipMalloc(&d_A, SIZE_A * sizeof(float));
    hipMalloc(&d_B, SIZE_B * sizeof(float));
    hipMalloc(&d_C, SIZE_C * sizeof(float));


    // Copy to device
    rc = hipMemcpy(d_A, &A[0], SIZE_A * sizeof(float), hipMemcpyHostToDevice);
    if (rc)
        std::cout << "error memcpy\n";
    hipMemcpy(d_B, &B[0], SIZE_B * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, SIZE_C * sizeof(float));

    // call the kernel
    matrixMultiplication(d_A, d_B, d_C, NA, MA, NB, MB);
    hipDeviceSynchronize();

    // copy memory back to host
    hipMemcpy(&C[0], d_C, SIZE_C * sizeof(float), hipMemcpyDeviceToHost);

    return C;
}