//
// Created by henri on 13/01/2020.
//

#include "matrix_add_scalar.hh"
#include "kernels/kernel_mat_op.hh"

double* mat_add_scalar(double* A, double scalar, int N, int M) {

    int SIZE = N*M;
    hipError_t rc = hipSuccess;
    // Allocate memory on the device
    double* d_A;
    double* d_B;
    auto* B = (double*)malloc(SIZE * sizeof(double));

    hipMalloc(&d_A, SIZE * sizeof(double));
    hipMalloc(&d_B, SIZE * sizeof(double));

    // Copy to device
    rc = hipMemcpy(d_A, &A[0], SIZE * sizeof(double), hipMemcpyHostToDevice);
    if (rc)
        std::cout << "error memcpy\n";
    hipMemset(d_B, 0, SIZE * sizeof(double));

    // call the kernel
    matrixAddScalar(d_A, d_B, scalar, N, M);
    hipDeviceSynchronize();

    // copy memory back to host
    hipMemcpy(&B[0], d_B, SIZE * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);

    return B;
}
