//
// Created by henri on 09/01/2020.
//

#include "matrix_conv.hh"
#include "kernels/kernel_mat_op.hh"

double* mat_conv(double* A, double* K, int NA, int MA, int NK, bool padding_valid) {

    if (NK % 2 == 0) {
        std::cerr << "shape error" << std::endl;
        return nullptr;
    }
    hipError_t rc = hipSuccess;

    int SIZE_A = NA*MA;
    int SIZE_K = NK*NK;



    // Allocate memory on the device
    double* d_A;
    double* d_B;
    double* d_C;

    hipMalloc(&d_A, SIZE_A * sizeof(double));
    hipMalloc(&d_B, SIZE_K * sizeof(double));


    // Copy to device
    rc = hipMemcpy(d_A, &A[0], SIZE_A * sizeof(double), hipMemcpyHostToDevice);
    if (rc)
        std::cout << "error memcpy\n";
    hipMemcpy(d_B, &K[0], SIZE_K * sizeof(double), hipMemcpyHostToDevice);

    if (!padding_valid) {
        int SIZE_C = SIZE_A;
        auto *C = (double *) malloc(SIZE_C * sizeof(double));
        hipMalloc(&d_C, SIZE_C * sizeof(double));
        hipMemset(d_C, 0, SIZE_C * sizeof(double));

        // call the kernel
        matrixConvSame(d_A, d_B, d_C, NA, MA, NK);
        hipDeviceSynchronize();

        // copy memory back to host
        hipMemcpy(&C[0], d_C, SIZE_C * sizeof(double), hipMemcpyDeviceToHost);

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return C;
    }

    else {
        int SIZE_C = (NA - 2*(NK/2)) * (MA - 2*(NK/2));
        auto *C = (double *) malloc(SIZE_C * sizeof(double));
        rc = hipMalloc(&d_C, SIZE_C * sizeof(double));
        if (rc)
            std::cout << "error malloc\n";
        rc = hipMemset(d_C, 0, SIZE_C * sizeof(double));
        if (rc)
            std::cout << "error memset\n";

        // call the kernel
        matrixConvValid(d_A, d_B, d_C, NA, MA, NK);
        hipDeviceSynchronize();

        // copy memory back to host
        rc = hipMemcpy(&C[0], d_C, SIZE_C * sizeof(double), hipMemcpyDeviceToHost);
        if (rc)
            std::cout << "error memcpy\n";

        hipFree(d_A);
        hipFree(d_B);
        hipFree(d_C);
        return C;
    }
}
