//
// Created by henri on 09/01/2020.
//

#include "matrix_conv.hh"
#include "kernels/kernel_mat_op.hh"

float* mat_conv(float* A, float* K, int NA, int MA, int NK, std::string padding) {

    if (NK % 2 == 0) {
        std::cerr << "shape error" << std::endl;
        return nullptr;
    }
    hipError_t rc = hipSuccess;

    int SIZE_A = NA*MA;
    int SIZE_K = NK*NK;



    // Allocate memory on the device
    float* d_A;
    float* d_B;
    float* d_C;

    hipMalloc(&d_A, SIZE_A * sizeof(float));
    hipMalloc(&d_B, SIZE_K * sizeof(float));


    // Copy to device
    rc = hipMemcpy(d_A, &A[0], SIZE_A * sizeof(float), hipMemcpyHostToDevice);
    if (rc)
        std::cout << "error memcpy\n";
    hipMemcpy(d_B, &K[0], SIZE_K * sizeof(float), hipMemcpyHostToDevice);

    if (padding == "same") {
        int SIZE_C = SIZE_A;
        auto *C = (float *) malloc(SIZE_C * sizeof(float));
        hipMalloc(&d_C, SIZE_C * sizeof(float));
        hipMemset(d_C, 0, SIZE_C * sizeof(float));

        // call the kernel
        matrixConvSame(d_A, d_B, d_C, NA, MA, NK);
        hipDeviceSynchronize();

        // copy memory back to host
        hipMemcpy(&C[0], d_C, SIZE_C * sizeof(float), hipMemcpyDeviceToHost);
        return C;
    }
    else if (padding == "valid") {
        int SIZE_C = (NA - 2*(NK/2)) * (MA - 2*(NK/2));
        auto *C = (float *) malloc(SIZE_C * sizeof(float));
        hipMalloc(&d_C, SIZE_C * sizeof(float));
        hipMemset(d_C, 0, SIZE_C * sizeof(float));

        // call the kernel
        matrixConvValid(d_A, d_B, d_C, NA, MA, NK);
        hipDeviceSynchronize();

        // copy memory back to host
        hipMemcpy(&C[0], d_C, SIZE_C * sizeof(float), hipMemcpyDeviceToHost);
        return C;
    }

    return nullptr;
}
