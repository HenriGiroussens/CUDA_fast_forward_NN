//
// Created by henri on 13/01/2020.
//

#include "matrix_avg_pooling.hh"
#include "kernels/kernel_mat_op.hh"

double* avg_pooling_2D(double* A, int N, int M, int strides, std::string padding) {
    int output_N = N / strides;
    int output_M = M / strides;
    if (padding == "same") {
        if (N%strides != 0)
            output_N++;
        if (M%strides != 0)
            output_M++;
    }
    int SIZE = output_N * output_M;
    hipError_t rc = hipSuccess;
    // Allocate memory on the device
    double* d_A;
    double* d_B;
    auto* B = (double*)malloc(SIZE * sizeof(double));

    hipMalloc(&d_A, N*M * sizeof(double));
    hipMalloc(&d_B, SIZE * sizeof(double));


    // Copy to device
    rc = hipMemcpy(d_A, &A[0], N*M* sizeof(double), hipMemcpyHostToDevice);
    if (rc)
        std::cout << "error memcpy\n";
    hipMemset(d_B, 0, SIZE * sizeof(double));

    // call the kernel
    matrixAvgPooling(d_A, d_B, N, M, output_N, output_M, strides);
    hipDeviceSynchronize();

    // copy memory back to host
    hipMemcpy(&B[0], d_B, N * sizeof(double), hipMemcpyDeviceToHost);

    return B;

}