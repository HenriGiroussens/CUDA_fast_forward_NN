//
// Created by henri on 09/01/2020.
//

#include "matrix_add.hh"

#include "kernels/kernel_mat_op.hh"


double* mat_add(double* A, double* B, int NA, int MA, int NB, int MB)
{
    if (NA != NB && MA != MB) {
        std::cerr << "shape error" << std::endl;
        return nullptr;
    }

    int SIZE = NA*MA;
    hipError_t rc = hipSuccess;
    // Allocate memory on the device
    double* d_A;
    double* d_B;
    double* d_C;
    auto* C = (double*)malloc(SIZE * sizeof(double));

    hipMalloc(&d_A, SIZE * sizeof(double));
    hipMalloc(&d_B, SIZE * sizeof(double));
    hipMalloc(&d_C, SIZE * sizeof(double));


    // Copy to device
    rc = hipMemcpy(d_A, &A[0], SIZE * sizeof(double), hipMemcpyHostToDevice);
    if (rc)
        std::cout << "error memcpy\n";
    hipMemcpy(d_B, &B[0], SIZE * sizeof(double), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, SIZE * sizeof(double));

    // call the kernel
    matrixAddition(d_A, d_B, d_C, NA, MA);
    hipDeviceSynchronize();

    // copy memory back to host
    hipMemcpy(&C[0], d_C, SIZE * sizeof(double), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return C;
}