//
// Created by henri on 09/01/2020.
//

#include "matrix_add.hh"

#include "kernels/kernel_mat_op.hh"


float* mat_add(float* A, float* B, int NA, int MA, int NB, int MB)
{
    if (NA != NB && MA != MB) {
        std::cerr << "shape error" << std::endl;
        return nullptr;
    }

    int SIZE = NA*MA;
    hipError_t rc = hipSuccess;
    // Allocate memory on the device
    float* d_A;
    float* d_B;
    float* d_C;
    auto* C = (float*)malloc(SIZE * sizeof(float));

    hipMalloc(&d_A, SIZE * sizeof(float));
    hipMalloc(&d_B, SIZE * sizeof(float));
    hipMalloc(&d_C, SIZE * sizeof(float));


    // Copy to device
    rc = hipMemcpy(d_A, &A[0], SIZE * sizeof(float), hipMemcpyHostToDevice);
    if (rc)
        std::cout << "error memcpy\n";
    hipMemcpy(d_B, &B[0], SIZE * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_C, 0, SIZE * sizeof(float));

    // call the kernel
    matrixAddition(d_A, d_B, d_C, NA, MA);
    hipDeviceSynchronize();

    // copy memory back to host
    hipMemcpy(&C[0], d_C, SIZE * sizeof(float), hipMemcpyDeviceToHost);

    return C;
}